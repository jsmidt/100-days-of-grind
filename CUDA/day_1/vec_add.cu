#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vecAdd(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n){
        C[i] = A[i] + B[i];
    }
}

int main() {
    int i;
    int n = 10;  // Size of the arrays
    float A_h[10], B_h[10], C_h[10];  // Declare arrays

    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // Initialize A_h to 1.0 and B_h to 2.0
    for (i = 0; i < n; i++) {
        A_h[i] = 1.0f;
        B_h[i] = 2.0f;
    }

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Call vecAdd
    // vecAdd(A_h, B_h, C_h, n);
    vecAdd<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Print the results
    printf("C_h array after vecAdd: ");
    for (i = 0; i < n; i++) {
        printf("%.1f ", C_h[i]);
    }
    printf("\n");

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}
