
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 4

__global__ void device_zip(int *a, int *b, int *out) {

        int index = threadIdx.x + blockIdx.x * blockDim.x;
        out[index] = a[index] + b[index];
}


int main(void) {
        int *a, *b, *out;
        int *d_a, *d_b, *d_out; // device copies of a, b, c
        int size = N * sizeof(int);

        // Alloc space for host copies of a, b, c and setup input values
        a = (int *)malloc(size);
        b = (int *)malloc(size);
        out = (int *)malloc(size);


        // Alloc space for device copies of a, b, c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_out, size);

        // Replicate arange
        for(int i=0;i<N;i++) {
                a[i] = i;
                b[i] = i;
        }

        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        int num_block = 1;
        int threads_per_block = 4;
        device_zip<<<num_block,threads_per_block>>>(d_a,d_b,d_out);

        // Copy result back to host
        hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

        for(int i=0;i<N;i++)
            printf(" %d + %d  = %d\n",  a[i], b[i], out[i]);

        free(a); free(b); free(out);
        hipFree(d_a); hipFree(d_b); hipFree(d_out);

        return 0;
}
